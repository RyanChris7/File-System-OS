﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
	fs->head = (u32) 0x0000FFFF;
	fs->tail= (u32) 0x0000FFFF;
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
 /* Implement open operation here */
	u32 hex = 0x0000FFFF;
	u32 bit8 = 0x000000FF;
	char n[20];
	char x;
	for(int i = 0;i < fs->FCB_ENTRIES;i++){
		for (int j = 0; j < fs->MAX_FILENAME_SIZE; j++) {
			x = fs->volume[fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE + j];
			if (x == '\0') break;
			n[j] = x;
		}
		
		hex = (fs->volume[fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE + x] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE + (x+1)];
	}

	if(hex == 0x0000FFFF){
		if(op == G_READ){
			printf("FILE IS NOT ON THE DISK!!");
			return (u32) 0x0000FFFF;
		}else {
			for(int i = 0; i < fs->FCB_ENTRIES;i++){
				if(((fs->volume[fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE + 22] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE + 23]) == hex){
					hex = i;
					break;
				}
			}
			u32 value1 = fs->SUPERBLOCK_SIZE + hex * fs->FCB_SIZE;
			fs->volume[value1 + 22] = 0;
			fs->volume[value1 + 23] = 0;
			for(int i = 0; i < fs->MAX_FILENAME_SIZE; i++){
				fs->volume[value1 + i] = s[i];
				if(s[i] == '\0')break;
			}

			u32 value2 = fs->SUPERBLOCK_SIZE + fs->head * fs->FCB_SIZE;
			if(fs->head == (u32) 0x0000FFFF) fs->tail = hex;
			else{
				fs->volume[value2 + 26] = hex & bit8;
				fs->volume[value2 + 27] = hex & bit8;
				fs->volume[value2 + 30] = hex & bit8;
				fs->volume[value2 + 31] = hex & bit8;
			}
			fs->volume[value1 + 24] = fs->head & bit8;
			fs->volume[value1 + 25] = fs->head & bit8;
			fs->volume[value1 + 28] = fs->head & bit8;
			fs->volume[value1 + 29] = fs->head & bit8;
			fs->head = hex;
		}
		return hex;
	}
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
}
__device__ void fs_gsys(FileSystem *fs, int op)
{
	/* Implement LS_D and LS_S operation here */
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
}
